#include "hip/hip_runtime.h"
# include "stranded.ch"
# include <iostream>
/*
__device__ int unsigned pos = 0;
__global__ void __update_window(boost::uint8_t * device, ub::matrix<qg::uni_part_t>::iterator1 it1, boost::uint16_t wsize[2], qg::coord_t * cam_pos)
{
	int unsigned x = pos;
	//device[x] = (* it1).rgba_colour.r;
	//device[x] = (* __uni)(cam_pos[0].xpos, cam_pos[0].ypos).rgba_colour.r;
	//device[x+1] = (* __uni)(cam_pos[0], cam_pos[0]).rgba_colour.g;
	//device[x+2] = (* __uni)(cam_pos[0], cam_pos[0]).rgba_colour.b;
	//device[x+3] = (* __uni)(cam_pos[0], cam_pos[0]).rgba_colour.a;
	//pos += 4;
}

static bool has_trig = false;
static boost::uint8_t * device;
//static int unsigned * pos;
static uni_part_t * host;
static int unsigned ssi;
*/
//void update_window(ub::matrix<ub::matrix<qg::uni_part_t> * __uni, boost::uint8_t * __win, boost::uint16_t wsize[2], int unsigned * pcount, qg::coord_t * cam_pos)
//{
/*
	if (has_trig == false)
	{
		ssi = (wsize[0] * wsize[1]);
		hipMalloc((void **)&device, (sizeof(boost::uint8_t) * (*pcount)));
		//daMalloc((void **)*host, (sizeof (uni_part_t) * (__uni.size1 () * __uni.size2 ())));
		//hipMalloc((void **)&pos, sizeof(int unsigned));
		has_trig = true;

	}
	
	//hipMemcpy( host, __uni.begin1(), s, hipMemcpyHostToDevice);
	//__update_window<<<1, ssi>>>(device, it1, wsize, cam_pos);
	//hipMemcpy(__win, device, (* pcount), hipMemcpyDeviceToHost );
*/
//}
__device__ static int unsigned pos = 0;
__global__ void __update_rwindow (qg::uni_part_t * __device_uni, boost::uint8_t * __device_win, boost::uint16_t * __w_size, boost::uint16_t * __u_size, qg::coord_t cam_pos)
{
	//int unsigned p = threadIdx.x + (blockIdx.x * __w_size[0];
	int unsigned pixel_location = threadIdx.x + (blockIdx.x * __w_size[0]);
	int unsigned unip_pos = threadIdx.x + (blockIdx.x * __u_size[0]);
	pixel_location = pixel_location * 4;
	__device_win[pixel_location] = __device_uni[unip_pos].rgba_colour.r;
	__device_win[pixel_location + 1] = __device_uni[unip_pos].rgba_colour.g;
	__device_win[pixel_location + 2] = __device_uni[unip_pos].rgba_colour.b;
	__device_win[pixel_location + 3] = __device_uni[unip_pos].rgba_colour.a;
/*
	if (pos > (((__w_size[0] * __w_size[1]) * 4) - 1))
		pos = 0;
	else
		pos += 4;
*/
}
//__device__ int unsigned oo = 0;
//__device__ static int unsigned dep = 0;

//__device__ int unsigned drop_down;
__global__ void __update_player (qg::uni_part_t * __device_uni, boost::uint16_t * __u_size, qg::player_info * main_player)
{
//	int unsigned *& drop_down = __drop_down;
	//int unsigned & __opl = *opl;
//	if (threadIdx.x == 0)
//	{
//		oo = 0;
//	}
        //int unsigned unip_pos = (threadIdx.x + main_player-> coords.xpos) + ((blockIdx.x + main_player-> coords.ypos) * __u_size[0]) + (oo * __u_size[0]);
	//unip_pos -= oo;
	//if (threadIdx.x == 0 && blockIdx.x == 0) __opl = 0;

	//if (threadIdx.x == 0) drop_down = 0;
		
	//drop_down = threadIdx.x;
	int unsigned uni_pos = (threadIdx.x + main_player-> coords.xpos) + ((blockIdx.x + main_player-> coords.ypos) * __u_size[0]);
	uni_pos += ((threadIdx.x * main_player->rotation) * __u_size[0]);
	uni_pos -= (blockIdx.x * main_player->rotation);
	
	__device_uni[uni_pos].rgba_colour.r = 255;
	__device_uni[uni_pos].rgba_colour.g = 255;
	__device_uni[uni_pos].rgba_colour.b = 255;
	__device_uni[uni_pos].rgba_colour.a = 255;

//	if (drop_down > 10)
//		drop_down = 0;
//rop_down ++;

//	if (threadIdx.x >= (main_player-> xlen)-3)
//		oo = 0;
//	else
//		oo ++;

}

__global__ void __clean_uni(qg::uni_part_t * __device_uni, boost::uint16_t * __u_size)
{
	int unsigned unip_pos = threadIdx.x + (blockIdx.x * __u_size[0]);
	__device_uni[unip_pos].rgba_colour.r = 0;
        __device_uni[unip_pos].rgba_colour.g = 0;
        __device_uni[unip_pos].rgba_colour.b = 0;
        __device_uni[unip_pos].rgba_colour.a = 0;
}

static boost::uint8_t __has_uw_been_init = false;
static boost::uint8_t * device_win;
static qg::uni_part_t * device_uni;
static int unsigned uni_bcount = 0;
static int unsigned ssi;
static qg::player_info * m_pinfo;
static boost::uint16_t * w_sz, * u_sz;
void update_window(qg::uni_part_t ** __universe, boost::uint8_t * __rwindow, boost::uint16_t __w_size[2], int unsigned * __p_count, qg::coord_t cam_pos, boost::uint16_t __u_size[2],
qg::player_info * main_player)
{
	if (__has_uw_been_init == false) {
		ssi = (__w_size[0] * __w_size[1]);
		hipMalloc((void **)&device_win, (sizeof(boost::uint8_t) * (*__p_count)));	
		hipMalloc((void **)&device_uni, (sizeof(qg::uni_part_t) * (__u_size[0] * __u_size[1])));
		__has_uw_been_init = true;
		hipMalloc((void **)&w_sz, sizeof(boost::uint16_t) * 2);
		hipMalloc((void **)&u_sz, sizeof(boost::uint16_t) * 2);
		hipMalloc((void **)& m_pinfo, sizeof(qg::player_info));	
		//ws[0] = __w_size[0]; ws[1] = __w_size[1];
		//us[0] = __u_size[0]; us[1] = __u_size[1];
		uni_bcount = (sizeof(qg::uni_part_t) * (__u_size[0] * __u_size[1])); 
		hipMemcpy(w_sz, __w_size, (sizeof(boost::uint16_t) * 2), hipMemcpyHostToDevice);
	        hipMemcpy(u_sz, __u_size, (sizeof(boost::uint16_t) * 2), hipMemcpyHostToDevice);
	}
//	 std::memset((* __universe), 0, uni_bcount);
	
	hipMemcpy(m_pinfo, main_player, sizeof(qg::player_info), hipMemcpyHostToDevice);
	hipMemcpy(device_uni, (* __universe), uni_bcount, hipMemcpyHostToDevice);
	__clean_uni<<<__u_size[1], __u_size[0]>>>(device_uni, u_sz);
	__update_player<<<main_player->ylen, main_player->xlen>>>(device_uni, u_sz, m_pinfo);
	__update_rwindow<<<__w_size[1], __w_size[0]>>>(device_uni, device_win, w_sz, u_sz, cam_pos);	

//	__update_player<<<main_player->ylen, main_player->xlen>>>(device_uni, u_sz, m_pinfo);
	hipMemcpy((* __universe), device_uni, uni_bcount, hipMemcpyDeviceToHost);
	hipMemcpy(__rwindow, device_win, (sizeof(boost::uint8_t) * (* __p_count)), hipMemcpyDeviceToHost);
}


void render_window()
{

}
